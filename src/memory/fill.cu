
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cstdint>

namespace nest {
namespace mc {
namespace memory {
namespace gpu {
    template <typename T, typename I>
    __global__
    void fill_kernel(T* v, T value, I n) {
        auto tid = threadIdx.x + blockDim.x*blockIdx.x;

        if(tid < n) {
            v[tid] = value;
        }
    }

    unsigned grid_dim(std::size_t n, unsigned block_dim) {
        return (n+block_dim-1)/block_dim;
    }

    void fill8(uint8_t* v, uint8_t value, std::size_t n) {
        unsigned block_dim = 192;
        fill_kernel<<<grid_dim(n, block_dim), block_dim>>>(v, value, n);
    };

    void fill16(uint16_t* v, uint16_t value, std::size_t n) {
        unsigned block_dim = 192;
        fill_kernel<<<grid_dim(n, block_dim), block_dim>>>(v, value, n);
    };

    void fill32(uint32_t* v, uint32_t value, std::size_t n) {
        unsigned block_dim = 192;
        fill_kernel<<<grid_dim(n, block_dim), block_dim>>>(v, value, n);
    };

    void fill64(uint64_t* v, uint64_t value, std::size_t n) {
        unsigned block_dim = 192;
        fill_kernel<<<grid_dim(n, block_dim), block_dim>>>(v, value, n);
    };
} // namespace gpu
} // namespace memory
} // namespace nest
} // namespace mc